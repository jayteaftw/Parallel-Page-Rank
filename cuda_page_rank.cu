#include "hip/hip_runtime.h"

#include "cuda_page_rank.h"

__global__
void cudaCalculate(int n, uns32*ptrs, uns32 *inds, flt32 *vals, flt32 *x, flt32 *y){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n){
        for(uns32 j = ptrs[i]; j < ptrs[i+1]; j++){
            y[i] += vals[j] *x[inds[j]];
        }  
    }
}

__global__
void cudaSwap(int n, flt32 *x, flt32 *y){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n){
        x[i] = y[i];
        y[i] = 0;
    }

}

void calculatePageRankCuda(SparseMatrix *adjM, flt32 *initPgRnkV, flt32 *finPgRnkV, uns32 N) {

    flt32 min_error = 0.0001;
    flt32 cur_error = min_error;
    cout<<"Cuda Check"<<endl;
    flt32 *d_x, *d_y, *vals;
    uns32 *inds, *ptrs;
    uns32 size = adjM->ptrs[N];
    hipMalloc(&d_x, N*sizeof(flt32)); 
    hipMalloc(&d_y, N*sizeof(flt32));
    hipMalloc(&ptrs, (N+1)*sizeof(uns32));
    hipMalloc(&vals, (size)*sizeof(flt32));
    hipMalloc(&inds, (size)*sizeof(uns32));

    cout<<"before cuda calc"<<endl;
    cout<<initPgRnkV[0]<<" "<<initPgRnkV[10]<<" "<<initPgRnkV[32]<<" "<<initPgRnkV[1000]<<" "<<endl; 
    cout<<finPgRnkV[0]<<" "<<finPgRnkV[10]<<" "<<finPgRnkV[32]<<" "<<finPgRnkV[1000]<<" "<<endl;   

    
    hipMemcpy(ptrs, adjM->ptrs, (N+1)*sizeof(uns32), hipMemcpyHostToDevice);
    hipMemcpy(inds, adjM->inds, size*sizeof(uns32), hipMemcpyHostToDevice);
    hipMemcpy(vals, adjM->vals, size*sizeof(flt32), hipMemcpyHostToDevice);

    for(int iter =0; iter < 10000; iter++){      
        
        hipMemcpy(d_x, initPgRnkV, N*sizeof(flt32), hipMemcpyHostToDevice);
        hipMemcpy(d_y, finPgRnkV, N*sizeof(flt32), hipMemcpyHostToDevice);
        uns32 block_size = 1024;

        for(int i = 0; i < 1000; i++)
        {
            cudaCalculate<<<(N+(block_size-1))/block_size, block_size>>>(N, ptrs, inds, vals, d_x, d_y);
            cudaSwap<<<N+(block_size-1)/block_size, block_size>>>(N, d_x, d_y);
        }

        cudaCalculate<<<(N+(block_size-1))/block_size, block_size>>>(N, ptrs, inds, vals, d_x, d_y);
        hipMemcpy(finPgRnkV, d_y, N*sizeof(flt32), hipMemcpyDeviceToHost);
        hipMemcpy(initPgRnkV, d_x, N*sizeof(flt32), hipMemcpyDeviceToHost);

        cout<<"After cuda calc"<<endl;
        cout<<"init "<<initPgRnkV[0]<<" "<<initPgRnkV[10]<<" "<<initPgRnkV[32]<<" "<<initPgRnkV[1000]<<" "<<endl; 
        cout<<"fin  "<<finPgRnkV[0]<<" "<<finPgRnkV[10]<<" "<<finPgRnkV[32]<<" "<<finPgRnkV[1000]<<" "<<endl<<endl;

        
        cur_error = 0;
        #pragma omp parallel for reduction(+:cur_error)
        for(uns32 idx = 0; idx < N; idx++){
            cur_error += abs(finPgRnkV[idx] - initPgRnkV[idx]);
            initPgRnkV[idx] = finPgRnkV[idx];
            finPgRnkV[idx] = 0;
        }
        cout<<"iter: "<<iter<<" error: "<<cur_error<<endl;
        if (cur_error < min_error){
            cout<<"iter: "<<iter<<" Final Error: "<< cur_error <<endl;
            break;
        }
        
    } 
    cout<<initPgRnkV[0]<<" "<<initPgRnkV[10]<<" "<<initPgRnkV[32]<<" "<<initPgRnkV[1000]<<" "<<endl; 

}